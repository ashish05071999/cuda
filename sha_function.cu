#include "hip/hip_runtime.h"
#include "sha_function.h"
#include <time.h>
#define word unsigned int 

/* f1 to f4 */

__device__ inline word f1( word x, word y, word z) { return ( ( x & y ) | ( ~x & z ) ); }
__device__ inline word f2( word x, word y, word z) { return ( x ^ y ^ z ); }
__device__ inline word f3( word x, word y, word z) { return ( ( x & y ) | ( x & z ) | ( y & z ) ); }
__device__ inline word f4( word x, word y, word z) { return ( x ^ y ^ z ); } 

/* SHA init values */

__constant__ word I1 = 0x67452301L;
__constant__ word I2 = 0xEFCDAB89L;
__constant__ word I3 = 0x98BADCFEL;
__constant__ word I4 = 0x10325476L;
__constant__ word I5 = 0xC3D2E1F0L;

/* SHA constants */

__constant__ word C1 = 0x5A827999L;
__constant__ word C2 = 0x6Ed9EBA1L;
__constant__ word C3 = 0x8F1BBCDCL;
__constant__ word C4 = 0xCA62C1D6L;

/* 32-bit rotate */

__device__ inline word ROT(word x,int n){ return ( ( x << n ) | ( x >> ( 32 - n ) ) ); }

/* main function */

#define CALC(n,i) temp =  ROT ( A , 5 ) + f##n( B , C, D ) +  W[i] + E + C##n  ; E = D; D = C; C = ROT ( B , 30 ); B = A; A = temp


__shared__ word * hash;

int main()
{
    char input[40], tmp[8]; 
    unsigned char * res;
    word hash[5];
    double time_tmp, time;

    // Init output and scan input

    printf("------------------------------------------------\n");
    printf("Welcome to a SHA-brute force programm using cuda\n");
    printf("------------------------------------------------\n\n");
    printf("Please enter your hash value: \t");

    
    scanf("%s", input);

    for(int i = 0; i < 5; i++)
    {
        for(int j = 0; j < 8; j++)
            tmp[j] = input[i * 8 + j];

        hash[i]=strtol(tmp,NULL,16);
    }
    
    printf("input verification: \t\t");
    for(int i = 0; i < 5; i++)
        printf("%X", hash[i]);

    printf("\n\n\n");

    res = (uc *) malloc(80);
    for(int i = 0; i < 10; i++)
        res[i] = 0;


    // Start calculation

    time = 0;   

    clock_t test = clock();
        
    printf("Checking for every possible 1-6 character password. \n\n");
   

    for(int i = 1; i < 7; i++)
    {
        printf("Execution started for string length %d\n.", i);
	// Function call
        start(hash, i, res);
        time_tmp = ( (double)clock() - test ) / CLOCKS_PER_SEC;
        time += time_tmp;
        printf("Finished. Time needed: %f\n", time_tmp);
        printf("Result: %s\n\n", (res[0] == 0 ? "No result found." : res));
	// If res != 0 (hash found) stop
	if(res[0] != 0) break;
        for(int j = 0; j < 10; j++)
            res[j] = 0;
    }
    // res still 0, no result
    if(res[0] == 0)
	printf("Unfortunately no valid hash was found :( \n Check your input character range. But maybe the password is too long?\n");
    else
	printf("Total execution time: %f \n", time);

    return 1;
}

void start(word * hash_tmp,  int length, unsigned char * res)
{
    unsigned char * buffer = 0;
    unsigned char * buffer_fill[10];
    cutilSafeCall ( hipMalloc((void** ) &buffer, 10 * sizeof(unsigned char)) );
    cutilSafeCall ( hipMalloc((void** ) &hash, 5 * sizeof(word)) );
    

    for(int i = 0; i <10; i++)
        buffer_fill[i] = 0x0; 
    
    hipMemcpy (hash, hash_tmp, 5 * sizeof(word), hipMemcpyHostToDevice);
    hipMemcpy (buffer, buffer_fill, 10 * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Call actual brute force kernel-function with 
    // - blocks: count of possible chars squared
    // - threads: possible chars
   
    smash<<<9025,95>>>(length, buffer, hash);

    hipMemcpy(res, buffer, 10 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    //hipMemcpy(debug, hash, 5 * sizeof(word), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
        printf( "Cuda error: %s.\n",  hipGetErrorString( err) );


    hipFree(buffer);
    hipFree(hash);
}


/*
 * kernel-function __global__ void smash(int, char, in)
 *
 * Initialize with count of possible chars squared as the block-num
 * and count of possible chars as the thread-num
 * With cx (where cx is char at position x of the tested word) the 
 * first 3 chars are set like:
 * 
 *   - c0: thread-num
 *   - c1: block-num / 95
 *   - c2: block-num % 95
 *
 * That guarantees every possible unique combination of the first
 * the chars.
 *
 * input:
 *   - length: length of the words 
 *   - buffer: buffer to write-back result, return value
 *   - hash: hash that needs to be decrypted
 *
*/

__global__ void smash(int length, unsigned char * buffer, word * hash)
{
    word h0,h1,h2,h3,h4;
    int higher = 126;
    int lower = 32;
    unsigned char input_cpy[10];
    int carry = 1;

    // load into register
    h0 = hash[0];
    h1 = hash[1];
    h2 = hash[2];
    h3 = hash[3];
    h4 = hash[4];

    if(length > 3)
       for(int i = 3; i < 10; i++)
           input_cpy[i] = lower;

    // init input_cpy
    input_cpy[0] = threadIdx.x + lower;
    if(length > 1)
		input_cpy[1] = (blockIdx.x / 95) + lower;
	if(length > 2)
	    input_cpy[2] = (blockIdx.x % 95) + lower;

    // Length for carry flag (break) if length < 3
    short int s = length < 3 ? length : 3;

    // value @length as a flag.
    // if != 0 break
    for(short int i = length; i < 10; i++)
            input_cpy[i] = 0;

    // Init words for SHA
    word W[80],A,B,C,D,E,temp;
 
    // calculate all possible charsets with the
    // given threadId, blockId and length
    while(input_cpy[length] == 0 && buffer[0] == 0) //@TODO || flag) 
    {
        // Calculate sha for given input.
        // DO THE SHA ------------------------------------------------------

        memInit(W, input_cpy, length);
        for(int i = 16; i < 80; i++)
            W[i] = ROT( ( W[i-3] ^ W[i-8] ^ W[i-14] ^ W[i-16] ) , 1 ); 

        A = I1;    B = I2;    C = I3;    D = I4;    E = I5;

        CALC(1,0);  CALC(1,1);  CALC(1,2);  CALC(1,3);  CALC(1,4);
        CALC(1,5);  CALC(1,6);  CALC(1,7);  CALC(1,8);  CALC(1,9);
        CALC(1,10); CALC(1,11); CALC(1,12); CALC(1,13); CALC(1,14);
        CALC(1,15); CALC(1,16); CALC(1,17); CALC(1,18); CALC(1,19);
        CALC(2,20); CALC(2,21); CALC(2,22); CALC(2,23); CALC(2,24);
        CALC(2,25); CALC(2,26); CALC(2,27); CALC(2,28); CALC(2,29);
        CALC(2,30); CALC(2,31); CALC(2,32); CALC(2,33); CALC(2,34);
        CALC(2,35); CALC(2,36); CALC(2,37); CALC(2,38); CALC(2,39);
        CALC(3,40); CALC(3,41); CALC(3,42); CALC(3,43); CALC(3,44);
        CALC(3,45); CALC(3,46); CALC(3,47); CALC(3,48); CALC(3,49);
        CALC(3,50); CALC(3,51); CALC(3,52); CALC(3,53); CALC(3,54);
        CALC(3,55); CALC(3,56); CALC(3,57); CALC(3,58); CALC(3,59);
        CALC(4,60); CALC(4,61); CALC(4,62); CALC(4,63); CALC(4,64);
        CALC(4,65); CALC(4,66); CALC(4,67); CALC(4,68); CALC(4,69);
        CALC(4,70); CALC(4,71); CALC(4,72); CALC(4,73); CALC(4,74);
        CALC(4,75); CALC(4,76); CALC(4,77); CALC(4,78); CALC(4,79);
    
        // That needs to be done, == with like (A + I1) =0 hash[0] 
        // is wrong all the time?!
        word tmp1, tmp2, tmp3, tmp4, tmp5;   
 
        tmp1 = A + I1;
        tmp2 = B + I2;
        tmp3 = C + I3;
        tmp4 = D + I4;
        tmp5 = E + I5;

        // if result was found, cpy to buffer
        if( tmp1 == h0 && 
            tmp2 == h1 &&
            tmp3 == h2 &&
            tmp4 == h3 &&
            tmp5 == h4 )
        { 
            buffer[0] = input_cpy[0];   
            buffer[1] = input_cpy[1];   
            buffer[2] = input_cpy[2];   
            buffer[3] = input_cpy[3];   
            buffer[4] = input_cpy[4];   
            buffer[5] = input_cpy[5];   
            buffer[6] = input_cpy[6];   
            buffer[7] = input_cpy[7];   
            buffer[8] = input_cpy[8];   
            buffer[9] = input_cpy[9];   
            
            break;
        }
        
        // adding new value
        // DO THE ADDITION ----------------------------------------------
    
        for(int i = s; i < 10; i++)
        {
            if(carry)
            {
                input_cpy[i] = input_cpy[i]+ 1;
                if(input_cpy[i] > higher)
                {
                    input_cpy[i] = lower;
                    carry = 1;
                } else 
                    carry = 0;
            } else 
                break;
        }

        carry = 1;

    }

}


/*
 * device function __device__ void memInit(uint, uchar, int)
 * 
 * Prepare word for sha-1 (expand, add length etc)
*/


__device__ void memInit(word * tmp, unsigned char input[], int length)
{

    int stop = 0;
    // reseting tmp
    for(int i = 0; i < 80; i++) tmp[i] = 0;

    // fill tmp like: message char c0,c1,c2,...,cn,10000000,00...000
    for(int i = 0; i < length; i+=4)
    {
        for(int j = 0; j < 4; j++)
            if(i + j < length)
                tmp[i/4] |= input[i+j] << (24-j * 8);
            else 
            {
                stop = 1;
                break;
            }
        if(stop)
            break;
    }
    tmp[length/4] |= 0x80 << (24-(length%4) * 8);     // Append 1 then zeros
    // Adding length as last value
    tmp[15] |= length * 8;
}
